#include <cudnn.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <hip/hip_fp16.h>

#define CHECK_CUDNN(expression)                             \
  {                                                         \
    cudnnStatus_t status = (expression);                    \
    if (status != CUDNN_STATUS_SUCCESS) {                   \
      std::cerr << "Error on line " << __LINE__ << ": "     \
                << cudnnGetErrorString(status) << std::endl;\
      std::exit(EXIT_FAILURE);                              \
    }                                                       \
  }

  int main(int argc, char *argv[]) {
    if (argc < 4) {
        std::cerr << "Usage: " << argv[0] << " input_size_of_first_dimension input_size_of_second_dimension time_iteration_size" << std::endl;
        return 1;
    }
    int H = 0;
    int W = 0;
    int T = 0;
    try {
      H = std::stoi(argv[1]);
      W = std::stoi(argv[2]);
      T = std::stoi(argv[3]);
    } catch (const std::invalid_argument &e) {
      std::cerr << "Invalid argument: cannot convert the parameter(s) to integer.\n";
      return 1;
    }

    std::cout << "Cudnn, " << "2d1r_half" << ", 1, " << H << ", " << W << ", " << T << ", ";

    cudnnHandle_t cudnn;
    CHECK_CUDNN(cudnnCreate(&cudnn));

    // int H = 10000;
    // int W = 10000;
    // int T = 10000;
    half *input_data_h;
    input_data_h = (half*)malloc(1 * 1 * H * W * sizeof(half));

    for (int i = 0; i < H * W; i++) {
        input_data_h[i] = 1.0f;
    }

    half *data[2];
    half *input_data;
    hipMalloc(&input_data, 1 * 1 * H * W * sizeof(half));
    hipMemcpy(input_data, input_data_h, 1 * 1 * H * W * sizeof(half), hipMemcpyHostToDevice);
    data[0] = input_data;

    cudnnTensorDescriptor_t input_descriptor;
    CHECK_CUDNN(cudnnCreateTensorDescriptor(&input_descriptor));
    CHECK_CUDNN(cudnnSetTensor4dDescriptor(input_descriptor,
                                           /*format=*/CUDNN_TENSOR_NHWC,
                                           /*dataType=*/CUDNN_DATA_HALF,
                                           /*batch_size=*/1,
                                           /*channels=*/1,
                                           /*image_height=*/H,
                                           /*image_width=*/W));

    half *filter_data_h;
    filter_data_h = (half*)malloc(1 * 1 * 3 * 3 * sizeof(half));

    for (int i = 0; i < 3 * 3; i++) {
        filter_data_h[i] = 0.1111f;
    }

    half *filter_data;
    hipMalloc(&filter_data, 1 * 1 * 3 * 3 * sizeof(half));
    hipMemcpy(filter_data, filter_data_h, 1 * 1 * 3 * 3 * sizeof(half), hipMemcpyHostToDevice);

    cudnnFilterDescriptor_t filter_descriptor;
    CHECK_CUDNN(cudnnCreateFilterDescriptor(&filter_descriptor));
    CHECK_CUDNN(cudnnSetFilter4dDescriptor(filter_descriptor,
                                           /*dataType=*/CUDNN_DATA_HALF,
                                           /*format=*/CUDNN_TENSOR_NCHW,
                                           /*out_channels=*/1,
                                           /*in_channels=*/1,
                                           /*kernel_height=*/3,
                                           /*kernel_width=*/3));

    cudnnConvolutionDescriptor_t convolution_descriptor;
    CHECK_CUDNN(cudnnCreateConvolutionDescriptor(&convolution_descriptor));
    CHECK_CUDNN(cudnnSetConvolution2dDescriptor(convolution_descriptor,
                                                /*pad_height=*/1,
                                                /*pad_width=*/1,
                                                /*vertical_stride=*/1,
                                                /*horizontal_stride=*/1,
                                                /*dilation_height=*/1,
                                                /*dilation_width=*/1,
                                                /*mode=*/CUDNN_CROSS_CORRELATION,
                                                /*computeType=*/CUDNN_DATA_HALF));
    CHECK_CUDNN(cudnnSetConvolutionMathType(convolution_descriptor, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));

    // 计算输出数据尺寸
    int batch_size{0}, channels{0}, height{0}, width{0};
    CHECK_CUDNN(cudnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                      input_descriptor,
                                                      filter_descriptor,
                                                      &batch_size,
                                                      &channels,
                                                      &height,
                                                      &width));

    half *output_data_h;
    output_data_h = (half*)malloc(batch_size * channels * height * width * sizeof(half));

    half *output_data;
    hipMalloc(&output_data, batch_size * channels * height * width * sizeof(half));
    data[1] = output_data;

    cudnnTensorDescriptor_t output_descriptor;
    CHECK_CUDNN(cudnnCreateTensorDescriptor(&output_descriptor));
    CHECK_CUDNN(cudnnSetTensor4dDescriptor(output_descriptor,
                                           /*format=*/CUDNN_TENSOR_NHWC,
                                           /*dataType=*/CUDNN_DATA_HALF,
                                           /*batch_size=*/batch_size,
                                           /*channels=*/channels,
                                           /*image_height=*/height,
                                           /*image_width=*/width));

    half alpha = 1.0f, beta = 0.0f;
    cudnnConvolutionFwdAlgo_t convolution_algorithm = CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    // CHECK_CUDNN(
    //     cudnnFindConvolutionForwardAlgorithm(cudnn,
    //                                         input_descriptor,
    //                                         filter_descriptor,
    //                                         convolution_descriptor,
    //                                         output_descriptor,
    //                                         CUDNN_CONVOLUTION_FWD_PREFER_FASTEST,
    //                                         /*memoryLimitInBytes=*/0,
    //                                         &convolution_algorithm));

    size_t workspace_bytes{0};
    CHECK_CUDNN(cudnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                        input_descriptor,
                                                        filter_descriptor,
                                                        convolution_descriptor,
                                                        output_descriptor,
                                                        convolution_algorithm,
                                                        &workspace_bytes));

    void* d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    for (int t = 0; t < T; t++) {
        CHECK_CUDNN(cudnnConvolutionForward(cudnn,
                                            &alpha,
                                            input_descriptor,
                                            data[t % 2],
                                            filter_descriptor,
                                            filter_data,
                                            convolution_descriptor,
                                            convolution_algorithm,
                                            d_workspace,
                                            workspace_bytes,
                                            &beta,
                                            output_descriptor,
                                            data[(t + 1) % 2]));
    }
    hipDeviceSynchronize() ;
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    // std::cout << "Time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;
    
    double secs = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1e6;
    // printf("GStencil/s = %f\n", ((double)H * W * T) / secs / 1e9);

    std::cout <<  std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << ", " << ((double)H * W) / secs / 1e9 * T << std::endl;

    hipMemcpy(output_data_h, output_data, batch_size * channels * height * width * sizeof(half), hipMemcpyDeviceToHost);




    cudnnDestroyTensorDescriptor(input_descriptor);
    cudnnDestroyTensorDescriptor(output_descriptor);
    cudnnDestroyFilterDescriptor(filter_descriptor);
    cudnnDestroyConvolutionDescriptor(convolution_descriptor);
    cudnnDestroy(cudnn);

    hipFree(input_data);
    hipFree(filter_data);
    hipFree(output_data);
    hipFree(d_workspace);

    return 0;
}
